#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>

#define CHECK_CUDA(call)                                                                       \
    do                                                                                         \
    {                                                                                          \
        hipError_t err = call;                                                                \
        if (err != hipSuccess)                                                                \
        {                                                                                      \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                \
        }                                                                                      \
    } while (0)

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

__global__ void crossAddKernel(float *deviceData, int *cnt)
{
    for (int strip = 1; strip < blockDim.x; strip *= 2)
    {
        if (threadIdx.x % (2 * strip) == 0)
        {
            deviceData[threadIdx.x] += deviceData[threadIdx.x + strip];
            atomicAdd(cnt, 1);
        }
        // NOTE: 如果没有这个，可能会出现当前层次没完成时先开始计算下一个层次的情况
        __syncthreads();
    }
}

__global__ void interAddKernel(float *deviceData, int *cnt)
{
    for (int strip = blockDim.x / 2; strip > 0; strip /= 2)
    {
        if (threadIdx.x < strip)
        {
            deviceData[threadIdx.x] += deviceData[threadIdx.x + strip];
            atomicAdd(cnt, 1);
        }
        __syncthreads();
    }
}

int main()
{

    // init
    float *hostData, *deviceData;
    int n = 1024000;
    int *cnt_a, *cnt_b, *da_cnt, *db_cnt;

    hostData = (float *)malloc(n * sizeof(float));
    cnt_a = (int *)malloc(sizeof(int));
    cnt_b = (int *)malloc(sizeof(int));
    *cnt_a = 0;
    *cnt_b = 0;

    float sum = 0.0f;
    for (int i = 0; i < n; i++)
    {
        hostData[i] = i;
    }

    struct timeval start, end;
    gettimeofday(&start, NULL);
    for (int i = 0; i < n; i++)
    {
        sum += i;
    }
    gettimeofday(&end, NULL);
    printf("EXCEPT: %.04f; TIME: %f\n", sum, (end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0);

    // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1g37d37965bfb4803b6d4e59ff26856356
    hipMalloc((void **)&deviceData, n * sizeof(float));
    hipMalloc(&da_cnt, sizeof(int));
    hipMemcpy(deviceData, hostData, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(da_cnt, cnt_a, sizeof(int), hipMemcpyHostToDevice);

    int BLOCK_DIM = 1024;
    int block_x = n / BLOCK_DIM; // 以之为整数倍打开线程，使至少一个线程运行 1024 个单元的计算。
    dim3 grid_dim(block_x, 1, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);

    // Cacluate 1: cross
    float cross_kernel_time;
    hipEvent_t start_cross, end_cross;
    hipEventCreate(&start_cross);
    hipEventCreate(&end_cross);
    hipEventRecord(start_cross, 0);

    crossAddKernel<<<grid_dim, block_dim>>>(deviceData, da_cnt);

    hipEventRecord(end_cross, 0);
    hipEventSynchronize(end_cross);
    hipEventElapsedTime(&cross_kernel_time, start_cross, end_cross);

    hipMemcpy(hostData, deviceData, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cnt_a, da_cnt, sizeof(int), hipMemcpyDeviceToHost);
    printf("CNT: %d; RESULT: %.04f;\n", *cnt_a, *hostData);

    for (int i = 0; i < n; i++)
    {
        hostData[i] = i;
        sum += i;
    }
    /// Cacluate 2: interleaving
    hipMalloc(&db_cnt, sizeof(int));
    hipMemcpy(deviceData, hostData, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db_cnt, cnt_b, sizeof(int), hipMemcpyHostToDevice);

    float inter_kernel_time;
    hipEvent_t start_inter, end_inter;
    hipEventCreate(&start_inter);
    hipEventCreate(&end_inter);
    hipEventRecord(start_inter, 0);

    interAddKernel<<<grid_dim, block_dim>>>(deviceData, db_cnt);

    hipEventRecord(end_inter, 0);
    hipEventSynchronize(end_inter);
    hipEventElapsedTime(&inter_kernel_time, start_inter, end_inter);

    hipMemcpy(hostData, deviceData, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cnt_b, db_cnt, sizeof(int), hipMemcpyDeviceToHost);
    printf("CNT: %d; RESULT: %.04f\n", *cnt_b, *hostData);
    CHECK_CUDA(hipDeviceSynchronize());

    free(hostData);
    free(cnt_a);
    free(cnt_b);
    printf("n = %d\n cross_time: %.4f\n inter_time: %.4f\n", n, cross_kernel_time, inter_kernel_time);
    return 0;
}
